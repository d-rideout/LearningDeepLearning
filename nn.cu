#include "hip/hip_runtime.h"
//#include <stdio.h>
//#define N 4

__global__ void mykernel(int *a, int *b, int *c) {
  int i = threadIdx.x;

  c[i] = a[i] + b[i];
}


int main(void) {
  int a[N], b[N], sum[N];
  int *da, *db, *dsum;
  size_t size = N*sizeof(int);

  hipMalloc(&da, size);
  hipMalloc(&db, size);
  hipMalloc(&dsum, size);

  for (int i=0; i<N; ++i) {
    a[i] = i;
    b[i] = 2*i+1;
  }

  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, size, hipMemcpyHostToDevice);

  mykernel<<<1,N>>>(da, db, dsum);

  //printf("Hello World! %d %d %d %d\n", a, b, c, d);

  hipMemcpy(sum, dsum, size, hipMemcpyDeviceToHost);
  //hipMemcpy(&d, dd, sizeof(int), hipMemcpyDeviceToHost);

  //printf("Hello World! %d %d %d %d\n", a, b, c, d);
  for (int i=0; i<N; ++i) printf("%d + %d = %d\n", a[i], b[i], sum[i]);

  hipFree(da);
  hipFree(db);
  hipFree(dsum);

  return 0;
}
