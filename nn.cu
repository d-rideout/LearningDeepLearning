#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "nn.h"

#define VERB 1
#define DEBUG 1
#define NTHREADS NBITS+1 // I don't need below
// (NBITS+1>NNEURONS ? NBITS+1 : NNEURONS) // max num inputs per neuron

/* Check for CUDA error */
 void cudaerr(const char *msg) {
   hipError_t cerr = hipGetLastError();
   //char *cerrst = hipGetErrorString(cerr);
   //or better:
   if (cerr) printf("CUDA error from %s : %s\n", msg, hipGetErrorString(cerr));
 }

#ifdef DEBUG
#define DI(i,j) ((NBITS+1)*i+j)
__global__ void learn(float *truth, float *in, float *a1, float *a2, float *out, float *debug) {
#else
__global__ void learn(float *truth, float *in, float *a1, float *a2, float *out) {
#endif
  __shared__ float temp[NNEURONS+1][NTHREADS];
  __shared__ float z[NNEURONS+1]; // FIX: +1 offset will be confusing!
  int ti = threadIdx.x; // index of dot product
  int ni = threadIdx.y; // index of neuron

  /* Forward Propagate */
  /* Layer 1 */

  /* Compute inner product */
  if (ni!=NNEURONS) temp[ni][ti] = a1[AI(ni,ti)] * in[ti];
#ifdef DEBUG
  debug[DI(ni,ti)] = temp[ni][ti];
#endif
  __syncthreads();
  if (!ti) {
    z[0] = 1.;
    // z[1..NNEURONS] z inner product on each neuron
    z[ni+1] = temp[ni][0];
    for (int i=1; i<NBITS+1; ++i) z[ni+1] += temp[ni][i];
  }
#ifdef DEBUG
  else z[1] = ti;
  //debug[DI(ni,ti)] = z[1]; correct inner product of 1st neuron
#endif
  __syncthreads();

  /* Layer 2 */
  // z[0] = 1 z[i] = z[neuron+1]

  /* Compute inner product */
  if (!ti) temp[ni][0] = a2[ni] * z[ni];
  __syncthreads();
  if (!ti && !ni) {
    *out = temp[0][0];
    for (int i=1; i<NNEURONS+1; ++i) *out += temp[i][0];
  }
#ifdef DEBUG
  //debug[DI(ni,ti)] = z[2]; // a2[ni]; //temp[1][0];
#endif
}


int main(void) {
  int i, j;
  const int nnum = 1<<NBITS;
  const int sf = sizeof(float);
  //  const int nthreads = NBITS+1>NNEURONS ? NBITS+1 : NNEURONS;

  /* MPI */
  //int nproc=1, myproc=0;

  /* GPU */
  int ngpu, mygpu;
  // hipGetDeviceProperties(hipDeviceProp_t *prop, int device)
  hipGetDeviceCount(&ngpu);
  hipGetDevice(&mygpu);
  printf("GPU %d of %d GPUs\n", mygpu, ngpu);
  // hipSetDevice(int device)

  /* Compute ground truth */
  const float y[] = {0,0,1.,1.,-1.,1.,-1.,1.,-1.,-1.,-1.,1.,-1.,1.,-1.,-1.};
  unsigned char num;
  float in[nnum][NBITS+1];
  for (num=2; num<nnum; ++num) {
    in[num][0] = 1.;
    for (i=1; i<NBITS+1; ++i) in[num][i] = num & 1<<(i-1) ? 1. : 0.;
  }
  if (VERB) for (num=2; num<nnum; ++num) {
    printf("%2d %3.0f : ", num, y[num]);
    printf("%2.f %2.f %2.f %2.f\n", in[num][0], in[num][1], in[num][2],
	   in[num][3]);
  }

  /* Define neural network */
  if (DEBUG) srand48(0);
  float *a1, *a2;
  a1 = (float *) malloc((NBITS+1)*NNEURONS*sf);
  //z1 = (float *) malloc(NNEURONS*sf);
  a2 = (float *) malloc((NNEURONS+1)*sf);
  for (i=0; i<NNEURONS; ++i) for (j=1; j<NBITS+1; ++j) {
    a1[AI(i,0)] = 0.; // start biases at 0
    a1[AI(i,j)] = 2*drand48()-1;
  }
  //for (i=0; i<NNEURONS; ++i) z1[i] = 2*drand48()-1;
  a2[0] = 0.;
  for (i=1; i<NNEURONS+1; ++i) a2[i] = 2*drand48()-1;
  float *out; // output value
  out = (float *) malloc(sf); // stack seems okay too

#ifdef DEBUG
  for (i=0; i<NNEURONS; ++i) {
    printf("\na1[%d] =", i);
    for (j=0; j<NBITS+1; ++j) printf("%10f", a1[AI(i,j)]);
  }
  printf("\na2    =");
  for (i=0; i<NNEURONS+1; ++i) printf("%10f", a2[i]);
  printf("\n");
  //float debug[NNEURONS][NBITS+1], *ddebug;
  float *debug, *ddebug;
  debug = (float *) malloc(sf*(NBITS+1)*(NNEURONS+1));
  //memset(debug, 'x', sf*(NBITS+1)*(NNEURONS+1));
  for (i=0; i<NNEURONS+1; ++i) {
    printf("nurn %d:", i);
    for (j=0; j<NBITS+1; ++j) {
      debug[DI(i,j)] = -9.;
      printf(" %9f", debug[DI(i,j)]);
    }
    printf("\n");
  }
  printf("-----------------------------------\n");
  hipMalloc(&ddebug, sf*(NBITS+1)*(NNEURONS+1));
#endif

  float *dtruth, *din, *da1, *da2, *dout;
  hipMalloc(&dtruth, sf);
  hipMalloc(&din, sf*(NBITS+1));
  hipMalloc(&da1, sf*(NBITS+1)*NNEURONS);
  hipMalloc(&da2, sf*(NNEURONS+1));
  hipMalloc(&dout, sf);
  
  /* Training loop */
  unsigned int sweep=0;
  unsigned char nwrong=1;
  
  while (nwrong) {
    if (sweep > MAX_SWEEP) return 1;
    nwrong = 0;
    ++sweep;

    /* Loop over data */
    // in principle should randomize order but seems like too much trouble
    for (num=2; num<1<<NBITS; ++num) {
      printf("num = %d :\n", num);
      //      for (i=0; i<

      /* Stage nn computation on GPU */
      hipMemcpy(dtruth, &y[num], sf, hipMemcpyHostToDevice);
      hipMemcpy(din, in[num], (NBITS+1)*sf, hipMemcpyHostToDevice);
      hipMemcpy(da1, a1, (NBITS+1)*NNEURONS*sf, hipMemcpyHostToDevice);
      hipMemcpy(da2, a2, (NNEURONS+1)*sf, hipMemcpyHostToDevice);

      /* Run the kernel */
      //dim3 threads(NTHREADS,NNEURONS);
      dim3 threads(NBITS+1,NNEURONS+1);
#ifdef DEBUG
      learn<<<1,threads>>>(dtruth, din, da1, da2, dout, ddebug);
#else
      learn<<<1,threads>>>(dtruth, din, da1, da2, dout);
#endif
      cudaerr("kernel");

      /* Copy results back to host */
      hipMemcpy(out, dout, sf, hipMemcpyDeviceToHost);
      cudaerr("out copy to host");
#ifdef DEBUG
      hipMemcpy(debug, ddebug, sf*(NBITS+1)*(NNEURONS+1), hipMemcpyDeviceToHost);
      cudaerr("debug copy to host");
      for (i=0; i<NNEURONS+1; ++i) {
	printf("nurn %d:", i);
	for (j=0; j<NBITS+1; ++j) printf(" %9f", debug[DI(i,j)]);
	printf("\n");
      }
#endif

      printf("NN output: %f\n", *out);
      //for (int i=0; i<N; ++i) printf("%d + %d = %d\n", a[i], b[i], sum[i]);
      return 2;

    } // loop over data
  } // training loop
  
  hipFree(dtruth);
  hipFree(din);
  hipFree(da1);
  hipFree(da1);
  hipFree(da1);

  return 0;
}
