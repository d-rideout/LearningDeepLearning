
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(int *a, int *b, int *c, int *d) {
  *d = threadIdx.x;

  *c = *a + *b;
}

int main(void) {
  int a, b, c=0, d;
  int *da, *db, *dc, *dd;

  hipMalloc(&da, sizeof(int));
  hipMalloc(&db, sizeof(int));
  hipMalloc(&dc, sizeof(int));
  hipMalloc(&dd, sizeof(int));

  a = 2;
  b = 9;

  hipMemcpy(da, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(db, &b, sizeof(int), hipMemcpyHostToDevice);

  mykernel<<<1,2>>>(da, db, dc, dd);

  printf("Hello World! %d %d %d %d\n", a, b, c, d);

  hipMemcpy(&c, dc, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&d, dd, sizeof(int), hipMemcpyDeviceToHost);

  printf("Hello World! %d %d %d %d\n", a, b, c, d);

  hipFree(da);
  hipFree(db);
  hipFree(dc);
  hipFree(dd);

  return 0;
}
